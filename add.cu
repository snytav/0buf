
#include <hip/hip_runtime.h>


__device__ double cuda_atomicAdd(double *address, double val)
{
    double assumed,old=*address;
    do {
        assumed=old;

        old=

        		__longlong_as_double
        		(
        		atomicCAS(
        				(unsigned long long int*)
        				address,
                    __double_as_longlong(assumed),
                    __double_as_longlong(val+assumed)));
    }while (assumed!=old);

    //printf("NEW ATOMIC ADD\n");

    return old;
}
