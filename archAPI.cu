#include "hip/hip_runtime.h"
/*
 * archAPI.cxx
 *
 *  Created on: Apr 10, 2018
 *      Author: snytav
 */
#include<stdlib.h>
#include<string.h>


#include "archAPI.h"



#ifdef __HIPCC__
int SetDevice(int n){return hipSetDevice(n);}
#else
int SetDevice(int n){return 0;}
#endif

#ifdef __HIPCC__
__device__
void AsyncCopy(double *dst,double *src,int n,int size)
{
	int j;
	j = n;
	if(j < size)
	{
	   dst[j] = src[j];
	}

}
#else
void AsyncCopy(double *dst,double *src,int n,int size){ memcpy(dst,src,n,size);}
#endif


#ifdef __HIPCC__
 int MemoryCopy(void* dst,void *src,size_t size,int dir)
{
//	int err = 0;


	hipMemcpyKind cuda_dir;

	if(dir == HOST_TO_DEVICE) cuda_dir = hipMemcpyHostToDevice;
	if(dir == HOST_TO_HOST) cuda_dir = hipMemcpyHostToHost;
	if(dir == DEVICE_TO_HOST) cuda_dir = hipMemcpyDeviceToHost;
	if(dir == DEVICE_TO_DEVICE) cuda_dir = hipMemcpyDeviceToDevice;



	return ((int)hipMemcpy(dst,src,size,cuda_dir));
}
#else
 int MemoryCopy(void* dst,void *src,size_t size,int dir);
#endif


#ifdef __HIPCC__
 int MemoryAllocate(void** dst,size_t size)
{
	hipMalloc(dst,size);
    return 0;
}
#else
 int MemoryAllocate(void** dst,size_t size);
#endif

#ifndef __HIPCC__
int GetDeviceMemory(size_t *m_free,size_t *m_total)
{
	*m_free = 0;
	*m_total = 0;
	return 0;
}
#endif

#ifndef __HIPCC__
int MemorySet(void *s, int c, size_t n)
{
	memset(s,c,n);
    return 0;
}
#endif


#ifndef __HIPCC__
int DeviceSynchronize()
{
    return 0;
}

#ifdef __HIPCC__
 int __host__ ThreadSynchronize()
{
	 return hipDeviceSynchronize();
}
#else
 int ThreadSynchronize()
 {
 	 return 0;
 }
#endif

 int getLastError()
{
	return 0;
}
#else
int getLastError()
{
	return (int)hipGetLastError();
}
#endif

#ifdef __HIPCC__
 __device__ void BlockThreadSynchronize()
 {
     __syncthreads();
 }
#else
 void BlockThreadSynchronize(){}
#endif

#ifdef __HIPCC__
__device__ double MultiThreadAdd(double *address, double val)
{
    double assumed,old=*address;
    do {
        assumed=old;
        old= __longlong_as_double(atomicCAS((unsigned long long int*)address,
                    __double_as_longlong(assumed),
                    __double_as_longlong(val+assumed)));
    }while (assumed!=old);

    *address += val;

    old = *address;

    return old;
}
#else
double MultiThreadAdd(double *address, double val){
#pragma omp critical
	*address += val;

}
#endif

#ifdef __HIPCC__
 const char *getErrorString(int err)
{
	return hipGetErrorString((hipError_t)err);
}
#else
const char *getErrorString(int err){return "";}

#endif


#ifdef __HIPCC__
 int GetDeviceMemory(size_t *m_free,size_t *m_total)
{
	return hipMemGetInfo(m_free,m_total);
}
#else
 int GetDeviceMemory(size_t *m_free,size_t *m_total){*m_free = -1; *m_total = -1;}
#endif


#ifdef __HIPCC__
int MemorySet(void *s, int c, size_t n)
{
	return (int)hipMemset(s,c,n);

//    return 0;
}
#else
int MemorySet(void *s, int c, size_t n)
{
	return memset(s,c,n);
}
#endif

int get_num_args(void **args)
{
	int i;
	for(i = 0;args[i] != NULL;i++);

	return i;
}

#ifndef __HIPCC__
dim3 threadIdx,blockIdx;
#endif

typedef void (*func_0)(void);

typedef void (*func_1)(void*);

typedef void (*func_2)(void*,void*);

typedef void (*func_3)(void*,void*,void*);

typedef void (*func_4)(void*,void*,void*,void*);

typedef void (*func_5)(void*,void*,void*,void*,void*);

typedef void (*func_6)(void*,void*,void*,void*,void*,void*);

typedef void (*func_7)(void*,void*,void*,void*,void*,void*,void*);

void call_with_args(const void *func, void **args)
{
	int num = get_num_args(args);



	if(num == 0)
    {
		func_0 f = (func_0)func;
	    f();
    }

	if(num == 1)
    {
	   func_1 f1 = (func_1)func;
	   f1(args[0]);
	}

	if(num ==2 )
	{
		func_2 f2 = (func_2)func;
        f2(args[0],args[1]);
	}

	if(num == 3)
	{
		func_3 f3 = (func_3)func;
        f3(args[0],args[1],args[2]);
	}

    if(num == 4)
    {
    	func_4 f4 = (func_4)func;
        f4(args[0],args[1],args[2],args[3]);
    }

    if(num == 5)
    {
    	func_5 f5 = (func_5)func;
        f5(args[0],args[1],args[2],args[3],args[4]);
    }

    if(num == 6)
    {
    	func_6 f6 = (func_6)func;
        f6(args[0],args[1],args[2],args[3],args[4],args[5]);
    }

	if(num == 7)
	{
		func_7 f7 = (func_7)func;
        f7(args[0],args[1],args[2],args[3],args[4],args[5],args[6]);
	}


}

int cudaLaunchKernel_onCPU(const void *func, dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream)
{

	for(int i = 0;i < gridDim.x;i++)
	{
		for(int l = 0;l < gridDim.y;l++)
		{
			for(int k = 0;k < gridDim.z;k++)
			{
#ifndef __HIPCC__
				blockIdx.x = i;
				blockIdx.y = l;
				blockIdx.z = k;
#endif

				for(int i1 = 0;i1 < blockDim.x;i1++)
				{
					for(int l1 = 0;l1 < blockDim.y;l1++)
					{
						for(int k1 = 0;k1 < blockDim.z;k1++)
						{
#ifndef __HIPCC__
							threadIdx.x = i1;
							threadIdx.x = l1;
							threadIdx.x = k1;
#endif
							call_with_args(func, args);
						}
					}
				}
			}
		}
	}

}

