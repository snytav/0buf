#include "hip/hip_runtime.h"
#include <math.h>
#include <string>
#include <stdio.h>

#include "archAPI.h"

#include <stdlib.h>
#include<string.h>

#include <sys/resource.h>
#include <stdint.h>

#include <sys/sysinfo.h>
#include <sys/time.h>

#include "particle.h"

//#include<hip/hip_runtime.h>

//struct sysinfo {
//       long uptime;             /* Seconds since boot */
//       unsigned long loads[3];  /* 1, 5, and 15 minute load averages */
//       unsigned long totalram;  /* Total usable main memory size */
//       unsigned long freeram;   /* Available memory size */
//       unsigned long sharedram; /* Amount of shared memory */
//       unsigned long bufferram; /* Memory used by buffers */
//       unsigned long totalswap; /* Total swap space size */
//       unsigned long freeswap;  /* swap space still available */
//       unsigned short procs;    /* Number of current processes */
//       unsigned long totalhigh; /* Total high memory size */
//       unsigned long freehigh;  /* Available high memory size */
//       unsigned int mem_unit;   /* Memory unit size in bytes */
//       char _f[20-2*sizeof(long)-sizeof(int)]; /* Padding for libc5 */
//   };

using namespace std;

int setPrintfLimit()
{
	size_t sizeP;

	printf("Particle size %lu %lu CurrentTensor %ld short %ld\n",sizeof(Particle),sizeof(Particle)/sizeof(double),sizeof(CurrentTensor),sizeof(char));

	hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);

	printf("printf default limit %lu \n",sizeP/1024/1024);

	sizeP *= 10000;
	hipDeviceSetLimit(hipLimitPrintfFifoSize, sizeP);

	hipDeviceGetLimit(&sizeP,hipLimitPrintfFifoSize);

	printf("printf limit set to %lu \n",sizeP/1024/1024);

	return 0;
}

double get_meminfo(void)
{
	FILE *f;
	char str[100];
	int  mem_free;
	double dmem;
   // return 0.0;

	system("free>&free_mem_out.dat");


	if((f = fopen("free_mem_out.dat","rt")) == NULL) return 0.0;

	fgets(str,100,f);
	fgets(str,100,f);

	mem_free = atoi(str + 30);

	dmem = (((double)mem_free)/1024)/1024;

	return dmem;

}

double get_meminfo1(void)
{
	double retval=0;
	char tmp[256]={0x0};
	/* note= add a path to meminfo like /usr/bin/meminfo
	   to match where meminfo lives on your system */
	FILE *shellcommand=popen("meminfo","r");
	while(fgets(tmp,sizeof(tmp),shellcommand)!=NULL)
	{
		if(memcmp(tmp,"Mem:",4)==0)
		{
			int	wordcount=0;
			std::string delimiter=" ";
			char *p=strtok(tmp,delimiter.c_str());
			while(*p)
			{
				wordcount++;
				if(wordcount==3) retval=atof(p);
			}
		}
	}
	pclose(shellcommand);
	return retval;
}


double CheckArraySilent	(double* a, double* dbg_a,int size)
	{
	   // Cell<Particle> c = (*AllCells)[0];
	    double diff = 0.0;

	    for(int n = 0;n < size;n++)
	    {
            diff += pow(a[n] - dbg_a[n],2.0);

//	        if(fabs(a[n] - dbg_a[n]) > TOLERANCE)
//		    {
//
//		       int3 i = c.getCellTripletNumber(n);
//
//     		}
	    }

	    return pow(diff/(size),0.5);
	}





void get_load_data_file_names(
		string & t_jxfile,
		string & t_jyfile,
		string & t_jzfile,
		string & t_d_jxfile,
		string & t_d_jyfile,
		string & t_d_jzfile,
		string & t_np_jxfile,
		string & t_np_jyfile,
		string & t_np_jzfile,
		string & t_qxfile,
		string & t_qyfile,
		string & t_qzfile,int nt)
{


    char d_exfile[100],d_eyfile[100],d_ezfile[100],d_hxfile[100],d_hyfile[100],d_hzfile[100];
    char d_0exfile[100],d_0eyfile[100],d_0ezfile[100];
    char jxfile[100],jyfile[100],jzfile[100];
    char np_jxfile[100],np_jyfile[100],np_jzfile[100];
    char np_exfile[100],np_eyfile[100],np_ezfile[100];
    char d_jxfile[100],d_jyfile[100],d_jzfile[100];
    char qxfile[100],qyfile[100],qzfile[100];
    char pfile[100],nextpfile[100];
//    char part_name[100];

    sprintf(qxfile,"dnqx%06d.dat",nt);
    sprintf(qyfile,"dnqy%06d.dat",nt);
    sprintf(qzfile,"dnqz%06d.dat",nt);



    sprintf(d_exfile,"dnex%06d.dat",2*nt-1);
    sprintf(d_eyfile,"dney%06d.dat",2*nt-1);
    sprintf(d_ezfile,"dnez%06d.dat",2*nt-1);

    sprintf(d_0exfile,"dnex%06d.dat",2*nt-2);
    sprintf(d_0eyfile,"dney%06d.dat",2*nt-2);
    sprintf(d_0ezfile,"dnez%06d.dat",2*nt-2);

    sprintf(d_hxfile,"dnhx%06d.dat",2*nt-1);
    sprintf(d_hyfile,"dnhy%06d.dat",2*nt-1);
    puts(d_hyfile);
    sprintf(d_hzfile,"dnhz%06d.dat",2*nt-1);

    sprintf(jxfile,"dnjx%06d.dat",2*nt);
    sprintf(jyfile,"dnjy%06d.dat",2*nt);
    sprintf(jzfile,"dnjz%06d.dat",2*nt);

    sprintf(d_jxfile,"npjx%06d.dat",2*nt);
    sprintf(d_jyfile,"npjy%06d.dat",2*nt);
    sprintf(d_jzfile,"npjz%06d.dat",2*nt);

    sprintf(np_jxfile,"npjx%06d.dat",2*nt);
    sprintf(np_jyfile,"npjy%06d.dat",2*nt);
    sprintf(np_jzfile,"npjz%06d.dat",2*nt);

    sprintf(np_exfile,"exlg%03d.dat",2*nt);
    sprintf(np_eyfile,"eylg%03d.dat",2*nt);
    sprintf(np_ezfile,"ezlg%03d.dat",2*nt);

    sprintf(pfile,    "part%06d000.dat",nt);
    sprintf(nextpfile,"part%06d000.dat",nt+2);


    t_jxfile =    jxfile;
    t_jyfile =    jyfile;
    t_jzfile =    jzfile;
    t_d_jxfile =  d_jxfile;
    t_d_jyfile =  d_jyfile;
    t_d_jzfile =  d_jzfile;
    t_np_jxfile = np_jxfile;
    t_np_jyfile = np_jyfile;
    t_np_jzfile = np_jzfile;
    t_qxfile =    qxfile;
    t_qyfile =    qyfile;
    t_qzfile =    qzfile;
}

void hipMalloc3D(double **X,double **Y,double**Z,int nx,int ny,int nz)
{
	hipMalloc(X,sizeof(double)*(nx+2)*(ny+2)*(nz+2));
	hipMalloc(Y,sizeof(double)*(nx+2)*(ny+2)*(nz+2));
	hipMalloc(Z,sizeof(double)*(nx+2)*(ny+2)*(nz+2));

}



void copyFieldsToGPU(
						double *d_Ex,double *d_Ey,double *d_Ez,
						double *d_Hx,double *d_Hy,double *d_Hz,
						double *d_Jx,double *d_Jy,double *d_Jz,
						double *d_npJx,double *d_npJy,double *d_npJz,
						double *d_Qx,double *d_Qy,double *d_Qz,
						double *Ex,double *Ey,double *Ez,
		        		double *Hx,double *Hy,double *Hz,
		        		double *Jx,double *Jy,double *Jz,
		        		double *npJx,double *npJy,double *npJz,
		                double *Qx,double *Qy,double *Qz,
		                int Nx,int Ny,int Nz
		)
{
	int err;

    err = MemoryCopy(d_Ex,Ex,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
    {
    	printf("1copyFieldsToGPU err %d %s \n",err,getErrorString(err));
    	exit(0);
    }
    err = MemoryCopy(d_Ey,Ey,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
    {
     	printf("2copyFieldsToGPU err %d %s \n",err,getErrorString(err));
    	exit(0);
    }

    err = MemoryCopy(d_Ez,Ez,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("3copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Hx,Hx,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("4copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }
    err = MemoryCopy(d_Hy,Hy,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("5copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }
    err = MemoryCopy(d_Hz,Hz,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("6copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Jx,Jx,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("7copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }
    err = MemoryCopy(d_Jy,Jy,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("8copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Jz,Jz,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("9copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_npJx,npJx,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("10copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_npJy,npJy,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("11copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_npJz,npJz,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("12copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Qx,Qx,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("13copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Qy,Qy,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("14copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }

    err = MemoryCopy(d_Qz,Qz,sizeof(double)*(Nx+2)*(Ny+2)*(Nz+2),HOST_TO_DEVICE);
    if(err != hipSuccess)
        {
         	printf("15copyFieldsToGPU err %d %s \n",err,getErrorString(err));
        	exit(0);
        }
}

void InitGPUFields(
		double **d_Ex,double **d_Ey,double **d_Ez,
		double **d_Hx,double **d_Hy,double **d_Hz,
		double **d_Jx,double **d_Jy,double **d_Jz,
		double **d_npJx,double **d_npJy,double **d_npJz,
        double **d_Qx,double **d_Qy,double **d_Qz,
        double *Ex,double *Ey,double *Ez,
		double *Hx,double *Hy,double *Hz,
		double *Jx,double *Jy,double *Jz,
		double *npJx,double *npJy,double *npJz,
		double *Qx,double *Qy,double *Qz,
		int Nx,int Ny,int Nz
        )
{
	hipMalloc3D(d_Ex,d_Ey,d_Ez,Nx,Ny,Nz);
	hipMalloc3D(d_Hx,d_Hy,d_Hz,Nx,Ny,Nz);
	hipMalloc3D(d_Jx,d_Jy,d_Jz,Nx,Ny,Nz);
	hipMalloc3D(d_npJx,d_npJy,d_npJz,Nx,Ny,Nz);
	hipMalloc3D(d_Qx,d_Qy,d_Qz,Nx,Ny,Nz);



    copyFieldsToGPU(
    		                        *d_Ex,*d_Ey,*d_Ez,
    								*d_Hx,*d_Hy,*d_Hz,
    								*d_Jx,*d_Jy,*d_Jz,
    								*d_npJx,*d_npJy,*d_npJz,
    								*d_Qx,*d_Qy,*d_Qz,
    								Ex,Ey,Ez,
    				        		Hx,Hy,Hz,
    				        		Jx,Jy,Jz,
    				        		npJx,npJy,npJz,
    				                Qx,Qy,Qz,
    				                Nx,Ny,Nz
    		);
}


